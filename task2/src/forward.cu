#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <algorithm>
#include <iostream>
#include "forward.h"

enum Activation {
    LeakyReLU,
    Tanh,
};

// __global__ void linear(
//     float *X_in,
//     float *X_out,
//     int n,
//     int d_in,
//     int d_out,
//     float *weights,
//     float *bias,
//     Activation activation,
//     float alpha)
// {
//     uint idx = blockIdx.x * blockDim.x + threadIdx.x;
//     if (idx >= n) {
//         return;
//     }

//     extern __shared__ float weights_s[];
//     if (threadIdx.x < d_in * d_out) {
//         weights_s[threadIdx.x] = weights[threadIdx.x];
//     }

//     __syncthreads();

//     float *x_in = X_in + idx * d_in;
//     float *x_out = X_out + idx * d_out;
//     for (int i = 0; i < d_out; ++i) {
//         float out = 0.;
//         int row = i * d_in;

//         for (int j = 0; j < d_in; ++j) {
//             out += x_in[j] * weights_s[row + j];
//         }
//         out += bias[i];

//         switch (activation) {
//         case Activation::LeakyReLU:
//             if (out < 0) {
//                 out = alpha * out;
//             }
//             break;
//         case Activation::Tanh:
//             out = tanh(out);
//         }

//         x_out[i] = out;
//     }
// }


struct Weights {
public:
    float **weights_host;
    float **weights_gpu;
    int n_layers;

    Weights(float **_weights_cpu, int *_sizes, int _n_layers) {
        n_layers = _n_layers;
        weights_host = (float **)malloc(_n_layers * sizeof(float *));
        for (int i = 0; i < 10; ++i) {
            checkCudaErrors(hipMalloc((void**)&weights_host[i], _sizes[i] * sizeof(float)));
            checkCudaErrors(hipMemcpy(
                weights_host[i],
                _weights_cpu[i],
                _sizes[i] * sizeof(float),
                hipMemcpyHostToDevice));
        }
        checkCudaErrors(hipMalloc((void**)&weights_gpu, 10 * sizeof(float *)));
        checkCudaErrors(hipMemcpy(
            weights_gpu,
            weights_host,
            _n_layers * sizeof(float *),
            hipMemcpyHostToDevice));
    }

    ~Weights() {
        for (int i = 0; i < n_layers; ++i) {
            checkCudaErrors(hipFree(weights_host[i]));
        }
        checkCudaErrors(hipFree(weights_gpu));
        free(weights_host);
    }
};


__device__ void linear_point(
    float *x_in,
    float *x_out,
    int d_in,
    int d_out,
    float *weights,
    float *bias,
    Activation activation,
    float alpha)
{
    for (int i = 0; i < d_out; ++i) {
        float out = 0.;
        int row = i * d_in;

        for (int j = 0; j < d_in; ++j) {
            out += x_in[j] * weights[row + j];
        }
        out += bias[i];

        switch (activation) {
        case Activation::LeakyReLU:
            if (out < 0) {
                out = alpha * out;
            }
            break;
        case Activation::Tanh:
            out = tanh(out);
        }

        x_out[i] = out;
    }
}


__device__ void forward_point(
    float *x_in,
    float *x_inner_1,
    float *x_inner_2,
    float *x_out,
    float **weights)
{
    linear_point(
        x_in,
        x_inner_1,
        3,
        32,
        weights[0],
        weights[1],
        Activation::LeakyReLU,
        0.1
    );

    float *tmp;
    for (int i = 0; i < 3; ++i) {
        linear_point(
            x_inner_1,
            x_inner_2,
            32,
            32,
            weights[2 + 2 * i],
            weights[3 + 2 * i],
            Activation::LeakyReLU,
            0.1
        );
        tmp = x_inner_1;
        x_inner_1 = x_inner_2;
        x_inner_2 = tmp;
    }

    linear_point(
        x_inner_1,
        x_out,
        32,
        1,
        weights[8],
        weights[9],
        Activation::Tanh,
        0.1
    );
}

__global__ void sphere_tracing(
    float *P,
    float *D,
    float *dist,
    float *X_inner_1,
    float *X_inner_2,
    float **weights,
    int W,
    int H)
{
    uint i = blockIdx.x * blockDim.x + threadIdx.x;
    uint j = blockIdx.y * blockDim.y + threadIdx.y;
    if (i >= H || j >= W) {
        return;
    }

    int idx = i * W + j;
    int p_idx = idx * 3;
    int i_idx = idx * 32;

    forward_point(
        P + p_idx,
        X_inner_1 + i_idx,
        X_inner_2 + i_idx,
        dist + idx,
        weights
    );

    float d = dist[idx];
    while (abs(d) > 1e-2 && d < 1) {
        P[p_idx] += D[p_idx] * d;
        P[p_idx + 1] += D[p_idx + 1] * d;
        P[p_idx + 2] += D[p_idx + 2] * d;

        forward_point(
            P + p_idx,
            X_inner_1 + i_idx,
            X_inner_2 + i_idx,
            dist + idx,
            weights
        );
        d = dist[idx];
    }
}


void forward(
    float *in,
    float *out,
    float **weights,
    int W,
    int H)
{
    float *inner1;
    float *inner2;
    float *in_gpu;
    float *out_gpu;
    // float *weights_gpu[10];
    int sizes[10];

    sizes[0] = 32 * 3;
    sizes[1] = 32;
    sizes[2] = 32 * 32;
    sizes[3] = 32;
    sizes[4] = 32 * 32;
    sizes[5] = 32;
    sizes[6] = 32 * 32;
    sizes[7] = 32;
    sizes[8] = 32;
    sizes[9] = 1;

    Weights weights_gpu_o(weights, sizes, 10);

    // for (int i = 0; i < 10; ++i) {
    //     checkCudaErrors(hipMalloc((void**)&weights_gpu[i], sizes[i] * sizeof(float)));
    //     checkCudaErrors(hipMemcpy(
    //         weights_gpu[i],
    //         weights[i],
    //         sizes[i] * sizeof(float),
    //         hipMemcpyHostToDevice));
    // }

    checkCudaErrors(hipMalloc((void**)&inner1, W * H * 32 * sizeof(float)));
    checkCudaErrors(hipMalloc((void**)&inner2, W * H * 32 * sizeof(float)));

    checkCudaErrors(hipMalloc((void**)&in_gpu, W * H * 3 * sizeof(float)));
    checkCudaErrors(hipMalloc((void**)&out_gpu, W * H * sizeof(float)));

    checkCudaErrors(hipMemcpy(
        in_gpu,
        in,
        W * H * 3 * sizeof(float),
        hipMemcpyHostToDevice));

	dim3 dimBlock2(32, 32);
	dim3 dimGrid2((H - 1) / 32 + 1, (W - 1) / 32 + 1);

    float *D = (float *)malloc(W * H * 3 * sizeof(float));
    for (int i = 0; i < W * H; ++i) {
        D[3 * i] = 0.0;
        D[3 * i + 1] = 0.;
        D[3 * i + 2] = -1.0;
    }
    float *D_gpu;
    checkCudaErrors(hipMalloc((void**)&D_gpu, W * H * 3 * sizeof(float)));
    checkCudaErrors(hipMemcpy(
        D_gpu,
        D,
        W * H * 3 * sizeof(float),
        hipMemcpyHostToDevice));

    sphere_tracing<<<dimGrid2, dimBlock2>>>(
        in_gpu,
        D_gpu,
        out_gpu,
        inner1,
        inner2,
        weights_gpu_o.weights_gpu,
        W,
        H
    );

    checkCudaErrors(hipDeviceSynchronize());

	// dim3 dimBlock(N_THREADS_PER_BLOCK);
	// dim3 dimGrid(((W * H) - 1) / N_THREADS_PER_BLOCK + 1);

    // linear<<<dimGrid, dimBlock, 3 * 32 * sizeof(float)>>>(
    //     in_gpu,
    //     inner1,
    //     W * H,
    //     3,
    //     32,
    //     weights_gpu[0],
    //     weights_gpu[1],
    //     Activation::LeakyReLU,
    //     0.1
    // );

    // for (int i = 0; i < 3; ++i) {
    //     linear<<<dimGrid, dimBlock, sizes[2 + 2 * i] * sizeof(float)>>>(
    //         inner1,
    //         inner2,
    //         W * H,
    //         32,
    //         32,
    //         weights_gpu[2 + 2 * i],
    //         weights_gpu[3 + 2 * i],
    //         Activation::LeakyReLU,
    //         0.1
    //     );
    //     std::swap(inner1, inner2);
    // }
    // linear<<<dimGrid, dimBlock, 32*sizeof(float)>>>(
    //     inner1,
    //     out_gpu,
    //     W * H,
    //     32,
    //     1,
    //     weights_gpu[8],
    //     weights_gpu[9],
    //     Activation::Tanh,
    //     0.
    // );

    // std::cout << 3 << std::endl;

    checkCudaErrors(hipMemcpy(
        out,
        out_gpu,
        W * H * sizeof(float),
        hipMemcpyDeviceToHost));
    checkCudaErrors(hipDeviceSynchronize());

    std::cout << 4 << std::endl;
}

void sphere_tracing(
    float* P,
    float* D,
    float eps,
    float **weights,
    int W,
    int H)
{
    float* dist;

    dist = (float*)malloc(W * H * sizeof(float));

    forward(
        P,
        dist,
        weights,
        W,
        H);

    int idx_0;
    int idx_1;
    for (int i = 0; i < H; ++i) {
        for (int j = 0; j < W; ++j) {
            idx_0 = i * W + j;
            idx_1 = idx_0 * 3;

            P[idx_1] += D[idx_1] * dist[idx_0];
            P[idx_1 + 1] += D[idx_1 + 1] * dist[idx_0 + 1];
            P[idx_1 + 2] += D[idx_1 + 2] * dist[idx_0 + 2];
        }
    }

    free(dist);
}
